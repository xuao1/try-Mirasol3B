#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 500000 // 数组大小，使得内核运行时间为几微秒
#define NSTEP 100 // 循环步数
#define NKERNEL 20 // 内核次数

// CUDA 内核函数定义
__global__ void shortKernel(float *out_d, float *in_d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) out_d[idx] = 1.23 * in_d[idx];
}

int main() {
    printf("Begin...\n");
    float *in_d, *out_d; // 设备端指针
    float *in_h, *out_h; // 主机端指针
    int size = N * sizeof(float); // 分配的内存大小

    // 分配主机端内存
    in_h = (float *)malloc(size);
    out_h = (float *)malloc(size);

    // 初始化输入数组
    for (int i = 0; i < N; i++) {
        in_h[i] = i;
    }

    // 分配设备端内存
    hipMalloc((void **)&in_d, size);
    hipMalloc((void **)&out_d, size);

    // 主机到设备的数据传输
    hipMemcpy(in_d, in_h, size, hipMemcpyHostToDevice);

    // 配置内核执行参数
    int threads = 512;
    int blocks = (N + threads - 1) / threads;

    // 创建 CUDA 流
    hipStream_t stream;
    hipStreamCreate(&stream);

    // 启动 CPU 计时器
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    printf("Exec...\n");

    // 内核执行循环
    bool graphCreated=false;
    hipGraph_t graph;
    hipGraphExec_t instance;

    for(int istep = 0; istep < NSTEP; istep++){
        if(!graphCreated){
            hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
            for(int ikrnl = 0; ikrnl < NKERNEL; ikrnl++){
                shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
            }
            hipStreamEndCapture(stream, &graph);
            hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
            graphCreated=true;
        }
        hipGraphLaunch(instance, stream);
        hipStreamSynchronize(stream);
    }

    printf("Done\n");

    // 结束 CPU 计时器
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Elapsed time: %f ms\n", milliseconds);

    // 设备到主机的数据传输
    hipMemcpy(out_h, out_d, size, hipMemcpyDeviceToHost);

    // 释放设备端内存
    hipFree(in_d);
    hipFree(out_d);

    // 释放主机端内存
    free(in_h);
    free(out_h);

    // 销毁 CUDA 流
    hipStreamDestroy(stream);

    return 0;
}
